#include "hip/hip_runtime.h"
#include "device.h"
#include "render/shared.h"
#include "render/shading.h"

#include <optix_device.h>

using namespace krr;
KRR_NAMESPACE_BEGIN

extern "C" __constant__ LaunchParamsGBuffer launchParams;

template <typename... Args>
KRR_DEVICE_FUNCTION void traceRay(OptixTraversableHandle traversable, Ray ray, float tMax,
								  int rayType, OptixRayFlags flags, Args &&...payload) {

	optixTrace(traversable, ray.origin, ray.dir, 0.f, tMax, 0.f, /* ray time val min max */
			   OptixVisibilityMask(255),						 /* all visible */
			   flags, rayType, 2,						/* ray type and number of types */
			   rayType,											 /* miss SBT index */
			   std::forward<Args>(payload)...);			/* (unpacked pointers to) payloads */
}

extern "C" __global__ void KRR_RT_CH(Primary)() {

}

extern "C" __global__ void KRR_RT_MS(Primary)() {

}

extern "C" __global__ void KRR_RT_RG(Primary)() {
	const Vector3ui launchIndex = optixGetLaunchIndex();
	const Vector2ui pixel		= {launchIndex[0], launchIndex[1]};
	const uint32_t pixelIndex	= pixel[0] + pixel[1] * launchParams.frameSize[0];

	PCGSampler pcgSampler;
	pcgSampler.setPixelSample(pixel, (uint32_t) launchParams.frameIndex);
	pcgSampler.advance(pixelIndex * 256);

	Sampler sampler = &pcgSampler;
	Ray ray = launchParams.cameraData.getRay(pixel, launchParams.frameSize, sampler);

	traceRay(launchParams.traversable, ray, KRR_RAY_TMAX, 0, OPTIX_RAY_FLAG_DISABLE_ANYHIT);
}

KRR_NAMESPACE_END